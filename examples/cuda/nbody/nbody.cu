#include "hip/hip_runtime.h"
#include "../../common/Stopwatch.hpp"

#include <hip/hip_runtime.h>
#include <fmt/format.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <llama/llama.hpp>
#include <random>
#include <string>
#include <utility>

using FP = float;

constexpr auto PROBLEM_SIZE = 64 * 1024; ///< total number of particles
constexpr auto SHARED_ELEMENTS_PER_BLOCK = 512;
constexpr auto STEPS = 5; ///< number of steps to calculate
constexpr auto ALLOW_RSQRT = true; // rsqrt can be way faster, but less accurate
constexpr FP TIMESTEP = 0.0001f;

constexpr auto THREADS_PER_BLOCK = 256;
constexpr auto AOSOA_LANES = 32; // coalesced memory access

// makes our life easier for now
static_assert(PROBLEM_SIZE % SHARED_ELEMENTS_PER_BLOCK == 0);
static_assert(SHARED_ELEMENTS_PER_BLOCK % THREADS_PER_BLOCK == 0);

constexpr FP EPS2 = 0.01;

// clang-format off
namespace tag
{
    struct Pos{};
    struct Vel{};
    struct X{};
    struct Y{};
    struct Z{};
    struct Mass{};
}

using Particle = llama::Record<
    llama::Field<tag::Pos, llama::Record<
        llama::Field<tag::X, FP>,
        llama::Field<tag::Y, FP>,
        llama::Field<tag::Z, FP>>>,
    llama::Field<tag::Vel, llama::Record<
        llama::Field<tag::X, FP>,
        llama::Field<tag::Y, FP>,
        llama::Field<tag::Z, FP>>>,
    llama::Field<tag::Mass, FP>
    // adding a padding element would nicely align a single Particle to 8 floats
    //, llama::Field<llama::NoName, FP>
>;

using ParticleJ = llama::Record<
    llama::Field<tag::Pos, llama::Record<
        llama::Field<tag::X, FP>,
        llama::Field<tag::Y, FP>,
        llama::Field<tag::Z, FP>>>,
    llama::Field<tag::Mass, FP>>;
// clang-format on

// using SharedMemoryParticle = Particle;
using SharedMemoryParticle = ParticleJ;

template <typename VirtualParticleI, typename VirtualParticleJ>
__device__ void pPInteraction(VirtualParticleI&& pi, VirtualParticleJ pj)
{
    auto dist = pi(tag::Pos()) - pj(tag::Pos());
    dist *= dist;
    const FP distSqr = EPS2 + dist(tag::X()) + dist(tag::Y()) + dist(tag::Z());
    const FP distSixth = distSqr * distSqr * distSqr;
    const FP invDistCube = ALLOW_RSQRT ? rsqrt(distSixth) : (1.0f / sqrt(distSixth));
    const FP sts = pj(tag::Mass()) * invDistCube * +TIMESTEP;
    pi(tag::Vel()) += dist * sts;
}

template <std::size_t ProblemSize, bool UseAccumulator, std::size_t BlockSize, int MappingSM, typename View>
__global__ void updateSM(View particles)
{
    // FIXME: removing this lambda makes nvcc 11 segfault
    auto sharedView = [] {
        constexpr auto sharedMapping = [] {
            constexpr auto arrayDomain = llama::ArrayDomain{BlockSize};
            if constexpr (MappingSM == 0)
                return llama::mapping::AoS{arrayDomain, SharedMemoryParticle{}};
            if constexpr (MappingSM == 1)
                return llama::mapping::SoA{arrayDomain, SharedMemoryParticle{}};
            if constexpr (MappingSM == 2)
                return llama::mapping::SoA{arrayDomain, SharedMemoryParticle{}, std::true_type{}};
            if constexpr (MappingSM == 3)
                return llama::mapping::AoSoA<decltype(arrayDomain), SharedMemoryParticle, AOSOA_LANES>{arrayDomain};
        }();

        llama::Array<std::byte*, decltype(sharedMapping)::blobCount> sharedMems{};
        boost::mp11::mp_for_each<boost::mp11::mp_iota_c<decltype(sharedMapping)::blobCount>>([&](auto i) {
            __shared__ std::byte sharedMem[sharedMapping.blobSize(i)];
            sharedMems[i] = &sharedMem[0];
        });
        return llama::View{sharedMapping, sharedMems};
    }();

    const auto ti = threadIdx.x + blockIdx.x * blockDim.x;
    const auto tbi = blockIdx.x;

    llama::One<Particle> pi;
    if constexpr (UseAccumulator)
        pi = particles(ti);
    for (std::size_t blockOffset = 0; blockOffset < ProblemSize; blockOffset += BlockSize)
    {
        LLAMA_INDEPENDENT_DATA
        for (auto j = tbi; j < BlockSize; j += THREADS_PER_BLOCK)
            sharedView(j) = particles(blockOffset + j);
        __syncthreads();

        LLAMA_INDEPENDENT_DATA
        for (auto j = std::size_t{0}; j < BlockSize; ++j)
        {
            if constexpr (UseAccumulator)
                pPInteraction(pi, sharedView(j));
            else
                pPInteraction(particles(ti), sharedView(j));
        }
        __syncthreads();
    }
    if constexpr (UseAccumulator)
        particles(ti) = pi;
}

template <std::size_t ProblemSize, bool UseAccumulator, typename View>
__global__ void update(View particles)
{
    const auto ti = threadIdx.x + blockIdx.x * blockDim.x;

    llama::One<Particle> pi;
    if constexpr (UseAccumulator)
        pi = particles(ti);
    LLAMA_INDEPENDENT_DATA
    for (auto j = std::size_t{0}; j < ProblemSize; ++j)
    {
        if constexpr (UseAccumulator)
            pPInteraction(pi, particles(j));
        else
            pPInteraction(particles(ti), particles(j));
    }
    if constexpr (UseAccumulator)
        particles(ti) = pi;
}

template <std::size_t ProblemSize, typename View>
__global__ void move(View particles)
{
    const auto ti = threadIdx.x + blockIdx.x * blockDim.x;
    particles(ti)(tag::Pos()) += particles(ti)(tag::Vel()) * +TIMESTEP;
}

void checkError(hipError_t code)
{
    if (code != hipSuccess)
        throw std::runtime_error(hipGetErrorString(code));
}

template <int Mapping, int MappingSM, bool UseAccumulator>
void run(std::ostream& plotFile, bool useSharedMemory)
try
{
    auto mappingName = [](int m) -> std::string {
        if (m == 0)
            return "AoS";
        if (m == 1)
            return "SoA";
        if (m == 2)
            return "SoA MB";
        if (m == 3)
            return "AoSoA" + std::to_string(AOSOA_LANES);
        if (m == 4)
            return "Split SoA";
    };
    auto title = "GM " + mappingName(Mapping);
    if (useSharedMemory)
        title += " SM " + mappingName(MappingSM);
    if (UseAccumulator)
        title += " Acc";
    std::cout << '\n' << title << '\n';

    auto mapping = [] {
        const auto arrayDomain = llama::ArrayDomain{PROBLEM_SIZE};
        if constexpr (Mapping == 0)
            return llama::mapping::AoS{arrayDomain, Particle{}};
        if constexpr (Mapping == 1)
            return llama::mapping::SoA{arrayDomain, Particle{}};
        if constexpr (Mapping == 2)
            return llama::mapping::SoA{arrayDomain, Particle{}, std::true_type{}};
        if constexpr (Mapping == 3)
            return llama::mapping::AoSoA<decltype(arrayDomain), Particle, AOSOA_LANES>{arrayDomain};
        if constexpr (Mapping == 4)
            return llama::mapping::Split<
                decltype(arrayDomain),
                Particle,
                llama::RecordCoord<1>,
                llama::mapping::SoA,
                llama::mapping::SoA,
                true>{arrayDomain};
    }();

    Stopwatch watch;

    auto hostView = llama::allocView(mapping);
    auto accView = llama::allocView(mapping, [](std::size_t size) {
        std::byte* p;
        checkError(hipMalloc(&p, size));
        return p;
    });

    watch.printAndReset("alloc");

    std::mt19937_64 generator;
    std::normal_distribution<FP> distribution(FP(0), FP(1));
    for (std::size_t i = 0; i < PROBLEM_SIZE; ++i)
    {
        llama::One<Particle> p;
        p(tag::Pos(), tag::X()) = distribution(generator);
        p(tag::Pos(), tag::Y()) = distribution(generator);
        p(tag::Pos(), tag::Z()) = distribution(generator);
        p(tag::Vel(), tag::X()) = distribution(generator) / FP(10);
        p(tag::Vel(), tag::Y()) = distribution(generator) / FP(10);
        p(tag::Vel(), tag::Z()) = distribution(generator) / FP(10);
        p(tag::Mass()) = distribution(generator) / FP(100);
        hostView(i) = p;
    }

    watch.printAndReset("init");

    hipEvent_t startEvent;
    hipEvent_t stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    auto start = [&] { checkError(hipEventRecord(startEvent)); };
    auto stop = [&] {
        checkError(hipEventRecord(stopEvent));
        checkError(hipEventSynchronize(stopEvent));
        float milliseconds = 0;
        checkError(hipEventElapsedTime(&milliseconds, startEvent, stopEvent));
        return milliseconds / 1000;
    };

    start();
    for (auto i = 0; i < accView.storageBlobs.rank; i++)
        checkError(hipMemcpy(
            accView.storageBlobs[i],
            hostView.storageBlobs[i].data(),
            mapping.blobSize(i),
            hipMemcpyHostToDevice));
    std::cout << "copy H->D " << stop() << " s\n";

    const auto blocks = PROBLEM_SIZE / THREADS_PER_BLOCK;

    double sumUpdate = 0;
    double sumMove = 0;
    for (std::size_t s = 0; s < STEPS; ++s)
    {
        start();
        if (useSharedMemory)
            updateSM<PROBLEM_SIZE, UseAccumulator, SHARED_ELEMENTS_PER_BLOCK, MappingSM>
                <<<blocks, THREADS_PER_BLOCK>>>(accView);
        else
            update<PROBLEM_SIZE, UseAccumulator><<<blocks, THREADS_PER_BLOCK>>>(accView);
        const auto secondsUpdate = stop();
        std::cout << "update " << secondsUpdate << " s\t";
        sumUpdate += secondsUpdate;

        start();
        move<PROBLEM_SIZE><<<blocks, THREADS_PER_BLOCK>>>(accView);
        const auto secondsMove = stop();
        std::cout << "move " << secondsMove << " s\n";
        sumMove += secondsMove;
    }
    if (!UseAccumulator)
        plotFile << std::quoted(title) << "\t" << sumUpdate / STEPS << '\t' << sumMove / STEPS << '\t';
    else
        plotFile << sumUpdate / STEPS << '\t' << sumMove / STEPS << '\n';

    start();
    for (auto i = 0; i < accView.storageBlobs.rank; i++)
        checkError(hipMemcpy(
            hostView.storageBlobs[i].data(),
            accView.storageBlobs[i],
            mapping.blobSize(i),
            hipMemcpyDeviceToHost));
    std::cout << "copy D->H " << stop() << " s\n";

    for (auto i = 0; i < accView.storageBlobs.rank; i++)
        checkError(hipFree(accView.storageBlobs[i]));
    checkError(hipEventDestroy(startEvent));
    checkError(hipEventDestroy(stopEvent));
}
catch (const std::exception& e)
{
    std::cerr << "Exception: " << e.what() << std::endl;
}

int main()
try
{
    std::cout << PROBLEM_SIZE / 1000 << "k particles (" << PROBLEM_SIZE * llama::sizeOf<Particle> / 1024 << "kiB)\n"
              << "Caching " << SHARED_ELEMENTS_PER_BLOCK << " particles ("
              << SHARED_ELEMENTS_PER_BLOCK * llama::sizeOf<SharedMemoryParticle> / 1024 << " kiB) in shared memory\n"
              << "Using " << THREADS_PER_BLOCK << " per block\n";
    int device = 0;
    hipGetDevice(&device);
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, device);
    std::cout << "Running on " << prop.name << " " << prop.sharedMemPerBlock / 1024 << "kiB SM\n";
    std::cout << std::fixed;

    std::ofstream plotFile{"nbody.tsv"};
    plotFile.exceptions(std::ios::badbit | std::ios::failbit);
    plotFile << "\"\"\t\"update\"\t\"move\"\t\"update with acc\"\t\"move with acc\"\n";

    using namespace boost::mp11;
    mp_for_each<mp_iota_c<5>>([&](auto i) {
        mp_for_each<mp_list_c<bool, false, true>>(
            [&](auto useAccumulator) { run<decltype(i)::value, 0, decltype(useAccumulator)::value>(plotFile, false); });
    });
    mp_for_each<mp_iota_c<5>>([&](auto i) {
        mp_for_each<mp_iota_c<4>>([&](auto j) {
            mp_for_each<mp_list_c<bool, false, true>>([&](auto useAccumulator) {
                run<decltype(i)::value, decltype(j)::value, decltype(useAccumulator)::value>(plotFile, true);
            });
        });
    });

    std::cout << "Plot with: ./nbody.sh\n";
    std::ofstream{"nbody.sh"} << fmt::format(
        R"(#!/usr/bin/gnuplot -p
set title "nbody CUDA {0}k particles"
set style data histograms
set style fill solid
set xtics rotate by 45 right
set key out top center maxrows 3
set yrange [0:*]
plot 'nbody.tsv' using 2:xtic(1) ti col, "" using 4 ti col
)",
        PROBLEM_SIZE / 1000);

    return 0;
}
catch (const std::exception& e)
{
    std::cerr << "Exception: " << e.what() << '\n';
}
