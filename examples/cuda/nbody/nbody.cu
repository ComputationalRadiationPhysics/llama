#include "hip/hip_runtime.h"
#include "../../common/Stopwatch.hpp"

#include <hip/hip_runtime.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <llama/llama.hpp>
#include <random>
#include <string>
#include <utility>

using FP = float;

constexpr auto PROBLEM_SIZE = 32 * 1024; ///< total number of particles
constexpr auto SHARED_ELEMENTS_PER_BLOCK = 1024;
constexpr auto STEPS = 5; ///< number of steps to calculate
constexpr FP TIMESTEP = 0.0001f;

constexpr FP ts = 0.0001;

constexpr auto THREADS_PER_BLOCK = 256;
constexpr auto AOSOA_LANES = 32; // coalesced memory access

// makes our life easier for now
static_assert(PROBLEM_SIZE % SHARED_ELEMENTS_PER_BLOCK == 0);
static_assert(SHARED_ELEMENTS_PER_BLOCK % THREADS_PER_BLOCK == 0);

constexpr FP EPS2 = 0.01;

// clang-format off
namespace tag
{
    struct Pos{};
    struct Vel{};
    struct X{};
    struct Y{};
    struct Z{};
    struct Mass{};
}

using Particle = llama::DS<
    llama::DE<tag::Pos, llama::DS<
        llama::DE<tag::X, FP>,
        llama::DE<tag::Y, FP>,
        llama::DE<tag::Z, FP>>>,
    llama::DE<tag::Vel, llama::DS<
        llama::DE<tag::X, FP>,
        llama::DE<tag::Y, FP>,
        llama::DE<tag::Z, FP>>>,
    llama::DE<tag::Mass, FP>>;
// clang-format on

template <typename VirtualParticleI, typename VirtualParticleJ>
__device__ void pPInteraction(VirtualParticleI pi, VirtualParticleJ pj)
{
    auto dist = pi(tag::Pos()) - pj(tag::Pos());
    dist *= dist;
    const FP distSqr = EPS2 + dist(tag::X()) + dist(tag::Y()) + dist(tag::Z());
    const FP distSixth = distSqr * distSqr * distSqr;
    const FP invDistCube = 1.0f / std::sqrt(distSixth);
    const FP sts = pj(tag::Mass()) * invDistCube * +TIMESTEP;
    pi(tag::Vel()) += dist * sts;
}

template <std::size_t ProblemSize, std::size_t BlockSize, int MappingSM, typename View>
__global__ void updateSM(View particles)
{
    // FIXME: removing this lambda makes nvcc 11 segfault
    auto sharedView = [] {
        auto sharedMapping = [] {
            const auto arrayDomain = llama::ArrayDomain{BlockSize};
            if constexpr (MappingSM == 0)
                return llama::mapping::AoS{arrayDomain, Particle{}};
            if constexpr (MappingSM == 1)
                return llama::mapping::SoA{arrayDomain, Particle{}};
            if constexpr (MappingSM == 2)
                return llama::mapping::AoSoA<decltype(arrayDomain), Particle, AOSOA_LANES>{arrayDomain};
        }();
        static_assert(decltype(sharedMapping)::blobCount == 1);
        constexpr auto sharedMemSize = llama::sizeOf<typename View::DatumDomain> * BlockSize;
        __shared__ std::byte sharedMem[sizeof(std::byte[sharedMemSize])];
        return llama::View{sharedMapping, llama::Array<std::byte*, 1>{&sharedMem[0]}};
    }();

    const auto ti = threadIdx.x + blockIdx.x * blockDim.x;
    const auto tbi = blockIdx.x;

    for (std::size_t blockOffset = 0; blockOffset < ProblemSize; blockOffset += BlockSize)
    {
        LLAMA_INDEPENDENT_DATA
        for (auto j = std::size_t{0}; j + ti < BlockSize; j += BlockSize)
            sharedView(j) = particles(blockOffset + j);
        __syncthreads();

        LLAMA_INDEPENDENT_DATA
        for (auto j = std::size_t{0}; j < BlockSize; ++j)
            pPInteraction(particles(ti), sharedView(j));
        __syncthreads();
    }
}

template <std::size_t ProblemSize, typename View>
__global__ void update(View particles)
{
    const auto ti = threadIdx.x + blockIdx.x * blockDim.x;

    LLAMA_INDEPENDENT_DATA
    for (auto j = std::size_t{0}; j < ProblemSize; ++j)
        pPInteraction(particles(ti), particles(j));
}

template <std::size_t ProblemSize, typename View>
__global__ void move(View particles)
{
    const auto ti = threadIdx.x + blockIdx.x * blockDim.x;
    particles(ti)(tag::Pos()) += particles(ti)(tag::Vel()) * +TIMESTEP;
}

void checkError(hipError_t code)
{
    if (code != hipSuccess)
        throw std::runtime_error(hipGetErrorString(code));
}

template <int Mapping, int MappingSM>
void run(const std::string& name, std::ostream& plotFile, bool useSharedMemory)
try
{
    auto mappingName = [](int m) -> std::string {
        if (m == 0)
            return "AoS";
        if (m == 1)
            return "SoA";
        if (m == 2)
            return "AoSoA" + std::to_string(AOSOA_LANES);
    };
    const auto title = name + " GlobalMemory " + mappingName(Mapping)
        + (useSharedMemory ? " SharedMemory " + mappingName(MappingSM) : "");
    std::cout << '\n' << title << '\n';

    auto mapping = [] {
        const auto arrayDomain = llama::ArrayDomain{PROBLEM_SIZE};
        if constexpr (Mapping == 0)
            return llama::mapping::AoS{arrayDomain, Particle{}};
        if constexpr (Mapping == 1)
            return llama::mapping::SoA{arrayDomain, Particle{}};
        if constexpr (Mapping == 2)
            return llama::mapping::AoSoA<decltype(arrayDomain), Particle, AOSOA_LANES>{arrayDomain};
    }();

    Stopwatch chrono;

    const auto bufferSize = mapping.getBlobSize(0);
    std::byte* accBuffer;
    checkError(hipMalloc(&accBuffer, bufferSize));

    chrono.printAndReset("alloc");

    auto hostView = llama::allocView(mapping);
    auto accView = llama::View<decltype(mapping), std::byte*>{mapping, llama::Array<std::byte*, 1>{accBuffer}};

    chrono.printAndReset("views");

    std::mt19937_64 generator;
    std::normal_distribution<FP> distribution(FP(0), FP(1));
    for (std::size_t i = 0; i < PROBLEM_SIZE; ++i)
    {
        auto temp = llama::allocVirtualDatumStack<Particle>();
        temp(tag::Pos(), tag::X()) = distribution(generator);
        temp(tag::Pos(), tag::Y()) = distribution(generator);
        temp(tag::Pos(), tag::Z()) = distribution(generator);
        temp(tag::Vel(), tag::X()) = distribution(generator) / FP(10);
        temp(tag::Vel(), tag::Y()) = distribution(generator) / FP(10);
        temp(tag::Vel(), tag::Z()) = distribution(generator) / FP(10);
        temp(tag::Mass()) = distribution(generator) / FP(100);
        hostView(i) = temp;
    }

    chrono.printAndReset("init");

    static_assert(hostView.storageBlobs.rank == 1);
    checkError(hipMemcpy(accBuffer, hostView.storageBlobs[0].data(), bufferSize, hipMemcpyHostToDevice));
    chrono.printAndReset("copy H->D");

    const auto blocks = PROBLEM_SIZE / THREADS_PER_BLOCK;

    double sumUpdate = 0;
    double sumMove = 0;
    for (std::size_t s = 0; s < STEPS; ++s)
    {
        if (useSharedMemory)
            updateSM<PROBLEM_SIZE, SHARED_ELEMENTS_PER_BLOCK, MappingSM><<<blocks, THREADS_PER_BLOCK>>>(accView);
        else
            update<PROBLEM_SIZE><<<blocks, THREADS_PER_BLOCK>>>(accView);
        checkError(hipDeviceSynchronize());
        sumUpdate += chrono.printAndReset("update", '\t');

        move<PROBLEM_SIZE><<<blocks, THREADS_PER_BLOCK>>>(accView);
        checkError(hipDeviceSynchronize());
        sumMove += chrono.printAndReset("move");
    }
    plotFile << std::quoted(title) << "\t" << sumUpdate / STEPS << '\t' << sumMove / STEPS << '\n';

    checkError(hipMemcpy(hostView.storageBlobs[0].data(), accBuffer, bufferSize, hipMemcpyDeviceToHost));
    chrono.printAndReset("copy D->H");

    checkError(hipFree(accBuffer));
}
catch (const std::exception& e)
{
    std::cerr << "Exception: " << e.what() << std::endl;
}

int main()
{
    std::cout << PROBLEM_SIZE / 1000 << "k particles (" << PROBLEM_SIZE * llama::sizeOf<Particle> / 1024 << "kiB)\n"
              << "Caching " << SHARED_ELEMENTS_PER_BLOCK << " particles ("
              << SHARED_ELEMENTS_PER_BLOCK * llama::sizeOf<Particle> / 1024 << " kiB) in shared memory\n"
              << "Using " << THREADS_PER_BLOCK << " per block\n";
    int device = 0;
    hipGetDevice(&device);
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, device);
    std::cout << "Running on " << prop.name << " " << prop.sharedMemPerBlock / 1024 << "kiB SM\n";

    std::ofstream plotFile{"nbody.tsv"};
    plotFile.exceptions(std::ios::badbit | std::ios::failbit);
    plotFile << "\"\"\t\"update\"\t\"move\"\n";

    run<0, 0>("LLAMA", plotFile, false);
    run<1, 0>("LLAMA", plotFile, false);
    run<2, 0>("LLAMA", plotFile, false);
    run<0, 0>("LLAMA", plotFile, true);
    run<0, 1>("LLAMA", plotFile, true);
    run<0, 2>("LLAMA", plotFile, true);
    run<1, 0>("LLAMA", plotFile, true);
    run<1, 1>("LLAMA", plotFile, true);
    run<1, 2>("LLAMA", plotFile, true);
    run<2, 0>("LLAMA", plotFile, true);
    run<2, 1>("LLAMA", plotFile, true);
    run<2, 2>("LLAMA", plotFile, true);

    std::cout << "Plot with: ./nbody.sh\n";
    std::ofstream{"nbody.sh"} << R"(#!/usr/bin/gnuplot -p
set style data histograms
set style fill solid
set xtics rotate by 45 right
set key out top center maxrows 3
set yrange [0:*]
plot 'nbody.tsv' using 2:xtic(1) ti col
)";

    return 0;
}